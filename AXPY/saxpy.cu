#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <Windows.h>

__global__ void saxpy(float *result, float *a, float *X, float *Y)
{
  // Linearize the thread arrangement
  int idx = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) 
          + threadIdx.z * (blockDim.y * blockDim.x) 
          + threadIdx.y * blockDim.x 
          + threadIdx.x;
  result[idx] = *a * X[idx] + Y[idx];
}

// Performs the single-precision AXPY operation (aX + Y), 
// where a is scalar and X,Y are vectors.
int main(int argc, char** argv)
{
  const int VECTOR_SIZE = 65536;
  const int VECTOR_BYTES = VECTOR_SIZE * sizeof(float);

  // Define a, X, and Y
  float h_a = 432.847;
  float *h_X = (float *)malloc(VECTOR_BYTES);
  float *h_Y = (float *)malloc(VECTOR_BYTES);
  float *h_result = (float *)malloc(VECTOR_BYTES); // result
  for (int i = 0; i < VECTOR_SIZE; i++)
  {
    h_X[i] = i;
    h_Y[i] = VECTOR_SIZE - i;
  }

  // Begin device operations

  // Allocate space in device memory for a, X, Y, and result
  float *d_a, *d_X, *d_Y, *d_result;
  hipMalloc((void **)&d_a, sizeof(float));
  hipMalloc((void **)&d_X, VECTOR_BYTES);
  hipMalloc((void **)&d_Y, VECTOR_BYTES);
  hipMalloc((void **)&d_result, VECTOR_BYTES);

  // Transfer a, X, Y to device
  hipMemcpy(d_a, &h_a, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_X, h_X, VECTOR_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_Y, h_Y, VECTOR_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_result, h_result, VECTOR_BYTES, hipMemcpyHostToDevice);

  // Start timer
  LARGE_INTEGER *startTime, *endTime, *freq;
  freq = (LARGE_INTEGER *)malloc(sizeof(LARGE_INTEGER));
  startTime = (LARGE_INTEGER *)malloc(sizeof(LARGE_INTEGER));
  endTime = (LARGE_INTEGER *)malloc(sizeof(LARGE_INTEGER));
  QueryPerformanceFrequency(freq);
  QueryPerformanceCounter(startTime);

  // Spawn threads
  saxpy<<<128, dim3(16,16,2)>>>(d_result, d_a, d_X, d_Y);

  // End timer
  QueryPerformanceCounter(endTime);
  __int64 elapsed = (*endTime).QuadPart - (*startTime).QuadPart;
  float elapsedTime = float(elapsed) / float((*freq).QuadPart);

  // Transfer result back to host
  hipMemcpy(h_result, d_result, VECTOR_BYTES, hipMemcpyDeviceToHost);
  
  // Display snippet of result
  printf("[%f,\n", h_result[0]);
  printf(" %f,\n", h_result[1]);
  printf(" %f,\n", h_result[2]);
  printf(" %f,\n", h_result[3]);
  printf(" %f,\n", h_result[4]);
  printf(" ...\n");
  printf(" %f]\n", h_result[VECTOR_SIZE - 1]);
  printf("\n");
  printf("Elapsed time (s): %f\n", elapsedTime);

  // Free objects
  hipFree(d_a);
  hipFree(d_result);
  hipFree(d_X);
  hipFree(d_Y);
  free(h_X);
  free(h_Y);
  free(h_result);

  exit(EXIT_SUCCESS);
}